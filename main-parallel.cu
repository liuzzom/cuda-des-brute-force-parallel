#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <time.h>
#include <iostream>
#define WS 8 // word size
#define CN 62 // characters number
#define DICT_SIZE 285465 // dictionary size
#define THREADS_NUM 512 // number of threads for each block


#include "hip/hip_runtime.h"
#include ""
#include "c_utils.h"
#include "des.h"
#include "des_utils.h"
#include "bit_utils.h"
#include "des_consts.h"
#include "des_kernel.h"
#include "cuda_utils.h"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* Check the return value of the CUDA runtime API call and exit
* the application if the call has failed.
*/
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "("
        << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}

// print an error message and terminate the program
void error(char *message){
	puts(message);
	exit(1);
}

// convert a string of 8 chars into an uint64_t
__device__ __host__ uint64_t strtouint64(char *string){
	uint64_t uword = 0;
	for(int i = 0; i < 8; i++){
		uint8_t uchar = (uint8_t) (int) string[i];
		uword += uchar;
		if(i < 7){
			uword<<=8;
		}
	}
	return uword;
}

__constant__ uint64_t c_target; // constant memory copy of target
__constant__ char c_charset[CN]; // constant memory copy of charset

// dictionary kernel
__global__ void dict_kernel(uint64_t *dictionary, uint64_t *result){
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	// check if the thread has some work to do
	if(index < DICT_SIZE){
		uint64_t word = dictionary[index];
		uint64_t hash_word = full_des_encode_block(word, word);
		if(hash_word == c_target){ // the thread found the solution
			*result = word;
			return;
		}
	}
}

__global__ void matrix_kernel(uint64_t *result, int offset){
	// thread coordinates
	int ix = blockIdx.x * blockDim.x + threadIdx.x; // blockIdx.x = 0 ix defines the column in the matrix
	int iy = (blockIdx.y * blockDim.y + threadIdx.y) + (offset * 16); // blockIdx.y = 0 iy defines the row in the matrix

	// shared control flag
	__shared__ int found;
	found = 0;

	__syncthreads();

	// check if the thread has some work to do
	if(ix < CN && iy < CN){
		// generated word
		char word[WS];
		word[0] = c_charset[iy]; // stays in registers
		word[1] = c_charset[ix]; // stays in registers

		for(int a = 0; a < CN; a++){
			for(int b = 0; b < CN; b++){
				for(int c = 0; c < CN; c++){
					for(int d = 0; d < CN; d++){
						for(int e = 0; e < CN; e++){
							for(int f = 0; f < CN; f++){
								if(found == 1){
									return;
								}

								word[2] = c_charset[a];
								word[3] = c_charset[b];
								word[4] = c_charset[c];
								word[5] = c_charset[d];
								word[6] = c_charset[e];
								word[7] = c_charset[f];

								// conversion and encryption
								uint64_t uword = strtouint64(word);
								uint64_t hash_word = full_des_encode_block(uword, uword);

								if(hash_word == c_target){
									*result = uword;
									found = 1;
									return;
								}
							}
						}
					}
				}
			}
		}
	}
	return;
}

int main(int argc, char **argv) {
	char *password = (char*)malloc(WS * sizeof(char)); // contains the target
	uint64_t upassword; // uint64_t version of the target
	uint64_t crypted_target; // uint64_t version of the crypted target

	char *curr_word = (char*)malloc(WS * sizeof(char)); // readed from dictionary or generated
	uint64_t uint_curr_word; // uint64_t version of curr_word

	FILE *dictionary; // dictionary
	uint64_t *h_dictionary = (uint64_t *) malloc(DICT_SIZE * sizeof(uint64_t)); // host copy of dictionary
	uint64_t *d_dictionary; // device copy of dictionary

	// charset
	char characters[CN] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z',
						   'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
						   '0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

	// decryption result
	uint64_t *result = (uint64_t *)malloc(sizeof(uint64_t));
	memset(result, 0, sizeof(uint64_t));
	uint64_t *d_result;

	// password to find
	password = "aaaaaazb";
	// verify if the user inserted eight characters password
	if((int)strlen(password) != 8){
		error("error: insert an eight characters password");
	}
	printf("target:%s\n", password);

	// conversion and encryption
	upassword = strtouint64(password);
	crypted_target = full_des_encode_block(upassword, upassword);
	printf("crypted target:");
	bits_print_grouped(crypted_target, 8, 64);

	// start counting clock cycles
	clock_t start_t = clock();

	puts("\nPhase 1: Try with dictionary");
	// open dictionary file
	puts("opening dictionary...");
	if((dictionary = fopen("/home/mauroliuzzo/cuda-workspace/des-brute-force-sequential/src/dictionary.txt", "r")) == NULL){
		error("error: dictionary not found");
	}
	puts("dictionary opened...");
	puts("");

	// dictionary import and converting
	puts("dictionary import and converting...");
	int i = 0;
	while(!feof(dictionary)){
		// import
		fscanf(dictionary, "%8s", curr_word);
		// conversion
		uint_curr_word = strtouint64(curr_word);
		// insert into the array
		h_dictionary[i] = uint_curr_word;
		i++;
	}
	//closing the file
	fclose(dictionary);
	puts("import/conversion done...");
	puts("");

	// gpu malloc and memset
	puts("gpu malloc and memset...");
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_dictionary, DICT_SIZE * sizeof(uint64_t)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_result, sizeof(uint64_t)));
	CUDA_CHECK_RETURN(hipMemset(d_result, 0, sizeof(uint64_t)));
	puts("malloc and memset done...");
	puts("");

	//gpu memcpy
	puts("gpu memcpy...");
	CUDA_CHECK_RETURN(hipMemcpy(d_dictionary, h_dictionary, DICT_SIZE * sizeof(uint64_t), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_target), &crypted_target, sizeof(uint64_t)));
	puts("gpu memcpy done...");
	puts("");

	// dictionary kernel launch
	puts("dictionary kernel launch...");
	int block_size = DICT_SIZE/THREADS_NUM + 1;
	dict_kernel<<<block_size, THREADS_NUM>>>(d_dictionary, d_result);

	// copying result
	CUDA_CHECK_RETURN(hipMemcpy(result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost));

	// check the result
	if(*result != 0x0000000000000000){
		printf("password found:");
		bits_print_grouped(*result, 8, 64);

		// gpu memory deallocation
		CUDA_CHECK_RETURN(hipFree(d_dictionary));
		CUDA_CHECK_RETURN(hipFree(d_result));

		// stop counting clock cycles and calculate elapsed time
		clock_t end_t = clock();
		clock_t total_t = (end_t - start_t);
		printf("Elapsed Time:%.3f seconds\n", (double)total_t/((double)CLOCKS_PER_SEC));

		return 0;
	}else{
		puts("password not in dictionary...");
	}

	// gpu memory deallocation
	CUDA_CHECK_RETURN(hipFree(d_dictionary));

	// Phase 2
	puts("\nPhase2: password generation");

	// copy the charset on device constant memory
	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_charset), characters, CN*sizeof(char)));

	// organizing threads
	dim3 block(64, 16);

	// matrix kernel launches
	for(int offset = 0; offset < 4; offset++){
		printf("%d° matrix kernel launch...\n", offset + 1);
		matrix_kernel<<<1, block>>>(d_result, offset);

		// copying result
		CUDA_CHECK_RETURN(hipMemcpy(result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost));

		// check the result
		if(*result != 0x0000000000000000){
			printf("password found:");
			bits_print_grouped(*result, 8, 64);

			// gpu memory deallocation
			CUDA_CHECK_RETURN(hipFree(d_result));

			// stop counting clock cycles and calculate elapsed time
			clock_t end_t = clock();
			clock_t total_t = (end_t - start_t);
			printf("Elapsed Time:%.3f seconds\n", (double)total_t/((double)CLOCKS_PER_SEC));

			return 0;
		}else{
			printf("password not found in %d matrix kernel...\n", offset + 1);
		}

	}
	return 0;
}
